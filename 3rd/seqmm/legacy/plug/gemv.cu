
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <iostream>

int bench(int m, int n) {
    int iter = 20;

    float *A, *x, *y;
    hipMalloc(&A, m * n * sizeof(float));
    hipMalloc(&x, n * sizeof(float));
    hipMalloc(&y, m * sizeof(float));

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.f;
    float beta = 0.f;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha, A, m, x, 1, &beta, y, 1);

    hipProfilerStart();
    hipEventRecord(start);
    for (int i = 0; i < iter; ++i) {
        hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha, A, m, x, 1, &beta, y, 1);
    }
    hipEventRecord(stop);
    hipProfilerStop();

    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << iter <<  " round hipsparseSpMM duration:" << ms << "ms" << std::endl;
    std::cout << "each round on average :" << ms / iter << "ms" << std::endl;

    hipFree(A);
    hipFree(x);
    hipFree(y);
    return 0;
}

int main() {
    int test_case[] = {8192, 1024, 8192, 8192, 24576, 8192, 32768, 8192, 8192, 32768};

    for (int i = 0; i < 5; ++i) {
        bench(test_case[i * 2], test_case[i * 2 + 1]);
    }

    return 0;
}



