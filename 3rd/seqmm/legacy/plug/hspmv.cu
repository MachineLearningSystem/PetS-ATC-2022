#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <hip/hip_runtime_api.h>

#include <vector>
#include <cstdlib>
#include <iomanip>
#include <chrono>
#include <iostream>

#include "half.hpp"

using half_float::half;

using thrust::device_vector;
using thrust::host_vector;

template<typename T>
void random_init(T* arr, size_t len, size_t sparsity){
    srand(45678);
    for (size_t i = 0; i < len; ++i) {
        if (rand() % 100 >= sparsity)
            arr[i] = rand() * 1.0 / RAND_MAX * 2.0f - 1.0f;
        else
            arr[i] = 0;
    }
}
template<typename DType>
void print_matrix(const DType *array, int row, int col) {
    // int j, k;
    // std::cout << "-----" << row << " x " << col << "-----" << std::endl;
    // std::cout.precision(4);
    // std::cout.flags(std::ios_base::fixed);
    // for(j = 0; j < row; ++j) {
    //     for(k = 0; k < col; ++k) {
    //         std::cout << std::setw(10) << array[j * col + k] - 0 << "\t";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << "===============" << std::endl;
}

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int bench(int m, int n){
    size_t sparsity = 99;

    int iter = 100;

    std::vector<half> A(m * n);
    std::vector<half> B(n);
    random_init<half>(A.data(), A.size(), sparsity);
    random_init<half>(B.data(), B.size(), 0);

    half *A_d, *B_d, *C_d;
    CHECK_CUDA(hipMalloc(&A_d, m * n * sizeof(half)));
    CHECK_CUDA(hipMalloc(&B_d, n * sizeof(half)));
    CHECK_CUDA(hipMalloc(&C_d, m * sizeof(float)));
    CHECK_CUDA(hipMemcpy(A_d, A.data(), m * n * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(B_d, B.data(), n * sizeof(half), hipMemcpyHostToDevice));

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    hipsparseDnMatDescr_t matA_dense;
    hipsparseDnVecDescr_t vecB, vecC;

    CHECK_CUSPARSE(hipsparseCreateDnMat(&matA_dense,
        m, n, n, A_d, HIP_R_16F, HIPSPARSE_ORDER_ROW));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecB, n, B_d, HIP_R_16F));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecC, m, C_d, HIP_R_32F));

    int *csrRowOffset;
    CHECK_CUDA(hipMalloc(&csrRowOffset, sizeof(int) * (m + 1)));

    hipsparseSpMatDescr_t matA;
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA,
        m, n, 0, csrRowOffset, NULL, NULL, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F));

    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(handle,
        matA_dense, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize));

    void *buffer;
    CHECK_CUDA(hipMalloc(&buffer, bufferSize));

    CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle,
        matA_dense, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, buffer));

    int64_t rows, cols, nnz;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matA, &rows, &cols, &nnz));
    printf("NNZ %ld\n", nnz);

    int *csrColIdx;
    half *csrData;
    CHECK_CUDA(hipMalloc(&csrColIdx, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc(&csrData, nnz * sizeof(half)));
    CHECK_CUSPARSE(hipsparseCsrSetPointers(matA, csrRowOffset, csrColIdx, csrData));

    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle,
        matA_dense, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, buffer));

    float alpha = 1.f;
    float beta = 0.f;
    hipsparseSpMVAlg_t spmv_alg = HIPSPARSE_SPMV_ALG_DEFAULT;

    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecB,
        &beta, vecC, HIP_R_32F, spmv_alg, &bufferSize));

    hipFree(buffer);
    CHECK_CUDA(hipMalloc(&buffer, bufferSize));

    CHECK_CUSPARSE(hipsparseSpMV(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecB,
        &beta, vecC, HIP_R_32F, spmv_alg, buffer));

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipProfilerStart());
    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < iter; ++i) {
        CHECK_CUSPARSE(hipsparseSpMV(handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecB,
            &beta, vecC, HIP_R_32F, spmv_alg, buffer));
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipProfilerStop());

    CHECK_CUDA(hipEventSynchronize(stop));
    float ms = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));
    std::cout << "True Sparity:" << 1 - nnz * 1.0 / ( m * n ) << std::endl;
    std::cout << iter <<  " round hipsparseSpMM duration:" << ms << "ms" << std::endl;
    std::cout << "each round on average :" << ms / iter << "ms" << std::endl;

    std::vector<half> C(m);
    CHECK_CUDA(hipMemcpy(C.data(), C_d, m * sizeof(half), hipMemcpyDeviceToHost));
    print_matrix<half>(C.data(), 1, 10);

    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matA_dense));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecB));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecC));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    CHECK_CUDA(hipFree(buffer));
    CHECK_CUDA(hipFree(A_d));
    CHECK_CUDA(hipFree(B_d));
    CHECK_CUDA(hipFree(C_d));

    return 0;
}

int main() {
    int test_case[] = {8192, 1024, 8192, 8192, 24576, 8192, 32768, 8192, 8192, 32768};

    for (int i = 0; i < 5; ++i) {
        bench(test_case[i * 2], test_case[i * 2 + 1]);
    }

    return 0;
}


