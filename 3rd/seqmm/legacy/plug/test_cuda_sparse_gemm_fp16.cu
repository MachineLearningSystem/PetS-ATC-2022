#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <vector>
#include <cstdlib>
#include <iomanip>
#include <chrono>
#include <iostream>

using thrust::device_vector;
using thrust::host_vector;

template<typename T>
void random_init(T* arr, size_t len, size_t sparsity){
    srand(45678);
    for (size_t i = 0; i < len; ++i) {
        if (rand() % 100 >= sparsity)
            arr[i] = rand() * 1.0 / RAND_MAX * 2.0f - 1.0f;
        else
            arr[i] = 0;
    }
}
template<typename DType>
void print_matrix(const DType *array, int row, int col) {
    int j, k;
    std::cout << "-----" << row << " x " << col << "-----" << std::endl;
    std::cout.precision(4);
    std::cout.flags(std::ios_base::fixed);
    for(j = 0; j < row; ++j) {
        for(k = 0; k < col; ++k) {
            float var = array[j * col + k];
            std::cout << std::setw(10) << var << "\t";
        }
        std::cout << std::endl;
    }
    std::cout << "===============" << std::endl;
}

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

int main(int argc, char const *argv[]){
    if (argc != 5) {
        std::cerr << "Usage: ./a.out m k n sparsity" << std::endl;
        exit(-1);
    }
    int m = atoi(argv[1]);
    int k = atoi(argv[2]);
    int n = atoi(argv[3]);
    size_t sparsity = atoi(argv[4]);
    std::vector<half> A(m * k);
    std::vector<half> B(k * n);
    random_init<half>(A.data(), A.size(), 0);
    random_init<half>(B.data(), B.size(), sparsity);

    device_vector<half> A_cuda(A);
    device_vector<half> B_cuda(B);
    device_vector<half> C_cuda(m * n);
    half* ptr_A = raw_pointer_cast(A_cuda.data());
    half* ptr_B = raw_pointer_cast(B_cuda.data());
    half* ptr_C = raw_pointer_cast(C_cuda.data());
    // gemm
    float alpha = 1.0f;
    float beta = 0.0f;
    bool transA_ = false;
    bool transB_ = true;
    hipblasOperation_t transA = transA_ ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transB = transB_ ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const int lda = transA_ ? m : k;
    const int ldb = transB_ ? k : n;
    const int ldc = n;

    // ---------------------------- cublas gemm ------------------------------ //
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, stream);
    hipblasGemmEx(handle, transB, transA, n, m, k, &alpha, ptr_B, HIP_R_16F, ldb,
            ptr_A, HIP_R_16F, lda, &beta, ptr_C, HIP_R_16F, ldc, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    hipEvent_t start,stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    int iter = 100;
    CHECK_CUDA(hipEventRecord(start,stream));
    for (int i = 0; i < iter; ++i) {
        hipblasGemmEx(handle, transB, transA, n, m, k, &alpha, ptr_B, HIP_R_16F, ldb,
                ptr_A, HIP_R_16F, lda, &beta, ptr_C, HIP_R_16F, ldc, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
    }
    CHECK_CUDA(hipEventRecord(stop,stream));
    CHECK_CUDA(hipEventSynchronize(stop));
    float milliseconds = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds,start,stop));
    std::cout << "hipblasGemmEx duration:" << milliseconds / iter << "ms" << std::endl;
    host_vector<half> C(C_cuda);
    // print_matrix<half>(A.data(), m, k);
    // print_matrix<half>(B.data(), k, n);
    print_matrix<half>(C.data(), 1, 10);
    if (handle) hipblasDestroy(handle);
    // ---------------------------- cublas gemm d------------------------------ //
    // sparse gemm
    hipsparseHandle_t sp_handle = 0;
    hipsparseCreate(&sp_handle);
    hipsparseSetStream(sp_handle, stream);
    // Perform matrix-matrix multiplication with the CSR-formatted matrix A
     // CUSPARSE APIs
    hipsparseSpMatDescr_t matB;
    hipsparseDnMatDescr_t matA, matC, matB_dense;
    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, k, m, lda, ptr_A,
                                        HIP_R_16F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB_dense, n, k, ldb, ptr_B,
                                        HIP_R_16F, HIPSPARSE_ORDER_ROW) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, n, m, ldc, ptr_C,
                                        HIP_R_16F, HIPSPARSE_ORDER_COL) )
    // Create sparse matrix B in CSR format
    int* csrRowPtr;
    int* csrColInd;
    half* csrVal;
    hipMalloc((void **)&csrRowPtr, sizeof(int) * (n + 1));
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, n, k, 0,
                                      csrRowPtr, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F) )
    //////////////// convert B to CSR
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseDenseToSparse_bufferSize(
                                        sp_handle, matB_dense, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseDenseToSparse_analysis(sp_handle, matB_dense, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )
    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp,
                                         &nnz) )

    // allocate CSR column indices and values
    CHECK_CUDA( hipMalloc((void**) &csrColInd, nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &csrVal,  nnz * sizeof(half)) )
    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE( hipsparseCsrSetPointers(matB, csrRowPtr, csrColInd, csrVal) )
    // execute Sparse to Dense conversion
    CHECK_CUSPARSE( hipsparseDenseToSparse_convert(sp_handle, matB_dense, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer) )

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 sp_handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matB, matA, &beta, matC, HIP_R_16F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    hipFree(dBuffer);
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(sp_handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matB, matA, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )
    CHECK_CUDA(hipEventRecord(start, stream));
    for (int i = 0; i < iter; ++i) {
        CHECK_CUSPARSE( hipsparseSpMM(sp_handle,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                     &alpha, matB, matA, &beta, matC, HIP_R_32F,
                                     HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )
    }
    CHECK_CUDA(hipEventRecord(stop, stream));
    CHECK_CUDA(hipEventSynchronize(stop));
    CHECK_CUDA(hipEventElapsedTime(&milliseconds,start,stop));
    std::cout << "True Sparity:" << 1 - nnz * 1.0 / ( k * n ) << std::endl;
    std::cout << "hipsparseSpMM duration:" << milliseconds / iter << "ms" << std::endl;
    host_vector<half> C_sparse(C_cuda);
    print_matrix<half>(C_sparse.data(), 1, 10);
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(sp_handle) )
    /////////////////////////////////////////////////////////////////////////////////////////
    std::cout << "M x K x N: " << m << 'x' << k << 'x' << n << std::endl;
    return 0;
}
